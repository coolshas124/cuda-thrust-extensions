
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/copy.h"
#include "thrust/reduce.h"
#include "thrust/sequence.h"
#include "thrust/generate.h"
#include "thrust/functional.h"

#include "big_integer.h"
#include <cstdlib>
#include <iostream>
#include <iterator>

void test_int_generate_and_reduce()
{
	std::cout<< "test generate and reduce with int" << std::endl;
	thrust::host_vector<thrust::big_integer<2>> h_v(10, 0xFFFF);
	
	std::cout << "Print host vector elements: ";
	for(int i = 0; i < h_v.size(); i++)
		std::cout << (unsigned int)h_v[i] << " ";
	std::cout << "\n";
	
	unsigned int sum = thrust::reduce(h_v.begin(), h_v.end());
	std::cout << "Reduce from host: " << sum << "\n";
	
	thrust::device_vector<thrust::big_integer<2>> d_v = h_v;
	sum = thrust::reduce(d_v.begin(), d_v.end());
	std::cout << "Reduce from device: " << sum << "\n";
}

int main()
{
	test_int_generate_and_reduce();
	
	system("pause");
	return 0;
}